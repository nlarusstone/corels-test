#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <gmp.h>

__global__ void Kernel(mp_limb_t *rule_a, mp_limb_t *rule_b, mp_limb_t *rule_c, int nentries)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i < nentries)
	   rule_c[i] = rule_a[i] & rule_b[i];
}

inline double timestamp()
{
	struct timeval now;
	gettimeofday(&now, 0);
	return now.tv_sec + now.tv_usec * 0.000001;
}

inline void randomize_vector(mpz_t v, int nsamples, gmp_randstate_t state)
{
    mpz_rrandomb(v, state, nsamples);
}

int main()
{
	double t0 = timestamp();

    gmp_randstate_t rand_state;
    gmp_randinit_mt(rand_state);

	mpz_t rule_a, rule_b, rule_c, cuda_rule_c;
    int nsamples = 100000000;
	mpz_init2(rule_a, nsamples);
    mpz_init2(rule_b, nsamples);
    mpz_init2(rule_c, nsamples);
    mpz_init2(cuda_rule_c, nsamples);
    randomize_vector(rule_a, nsamples, rand_state);
    randomize_vector(rule_b, nsamples, rand_state);

    double t1 = timestamp();

    int nentries = abs(rule_a->_mp_size);

    mp_limb_t *d_rule_a, *d_rule_b, *d_rule_c;
    size_t s = nentries * sizeof(mp_limb_t);

	hipMalloc(&d_rule_a, s);
	hipMalloc(&d_rule_b, s);
	hipMalloc(&d_rule_c, s);

	hipMemcpy(d_rule_a, rule_a->_mp_d, s, hipMemcpyHostToDevice);
	hipMemcpy(d_rule_b, rule_b->_mp_d, s, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (nentries + threadsPerBlock - 1) / threadsPerBlock;
	dim3 blocks(blocksPerGrid, 1, 1);
	dim3 threads(threadsPerBlock, 1, 1);

	double t2 = timestamp();

	Kernel<<<blocks, threads>>>(d_rule_a, d_rule_b, d_rule_c, nentries);
	hipDeviceSynchronize();

	double t3 = timestamp();

	hipMemcpy(cuda_rule_c->_mp_d, d_rule_c, s, hipMemcpyDeviceToHost);

	hipFree(d_rule_a);
	hipFree(d_rule_b);
	hipFree(d_rule_c);

    double t4 = timestamp();

    mpz_and(rule_c, rule_a, rule_b);

    double t5 = timestamp();

    cuda_rule_c->_mp_size = rule_c->_mp_size;

	mpz_clear(rule_a);
	mpz_clear(rule_b);
	mpz_clear(rule_c);
    mpz_clear(cuda_rule_c);

    double t6 = timestamp();

    if(mpz_cmp(rule_c, cuda_rule_c) == 0) {
        printf("Outputs match!\n");
    }

	printf("\n\nSetup time: %.8f\n", t1 - t0);
    printf("CUDA kernel run time: %.8f\n", t3 - t2);
    printf("CUDA total run time: %.8f\n", t4 - t1);
    printf("GMP total run time: %.8f\n", t5 - t4);
    printf("Total time: %.8f\n", t6 - t0);

	return 0;
}
